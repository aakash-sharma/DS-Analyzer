/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <assert.h>
#define DEBUG
#define GPUS 2

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

void profileCopies(float        *h_a, 
                   float        *h_b, 
                   float        *d, 
                   unsigned int  n,
                   unsigned int  iterations)
{
  unsigned int bytes = n * sizeof(float);

  // events for timing
  hipEvent_t startEvent, stopEvent; 
  
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  
  checkCuda( hipMemcpy(d[0], h_a, bytes, hipMemcpyHostToDevice) );
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < iterations; i++)
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  float time;
  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 * iterations / time);

  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < iterations; i++)
  	checkCuda( hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 * iterations/ time);

  for (int i = 0; i < n; ++i) {
    if (h_a[i] != h_b[i]) {
      printf("*** transfers failed ***");
      break;
    }
  }

  // clean up events
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
}

int main()
{
  unsigned int nElements = 4*1024*1024;
  const unsigned int bytes = nElements * sizeof(float);

  // host arrays
  float *h_aPageable, *h_bPageable;   

  // device array
  float *d_a[GPUS];

  // allocate and initialize
  h_aPageable = (float*)malloc(bytes);                    // host pageable
  h_bPageable = (float*)malloc(bytes);                    // host pageable

  for (int i = 0; i < GPUS; i++) 
  { 
    checkCuda (hipSetDevice(i) );
    checkCuda( hipMalloc((void**)&d_a[i], bytes) );           // device
  }

  for (int i = 0; i < nElements; ++i) h_aPageable[i] = i;      
  memset(h_bPageable, 0, bytes);

  // output device info and transfer size
  hipDeviceProp_t prop[GPUS];
  for (int i = 0; i < GPUS; i++) 
  {
    checkCuda( hipSetDevice(i) );
    checkCuda( hipGetDeviceProperties(&prop[i], i) );
    printf("\nDevice %d: %s\n", i, prop.name);
  }

  printf("Transfer size (MB): %d\n", bytes / (1024 * 1024));

  // perform copies and report bandwidth
  profileCopies(h_aPageable, h_bPageable, d_a, nElements, 100000);

  printf("\n");

  // cleanup
  for (int i = 0; i < GPUS; i++)  
  {
    checkCuda (hipSetDevice(i) );
    hipFree(d_a[i]);
  }

  free(h_aPageable);
  free(h_bPageable);

  return 0;
}
